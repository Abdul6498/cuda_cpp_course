﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <cstring>
#include <cassert>
#include <chrono>

__global__ void add(int* a, int *b, int *c, int size) {

	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		c[gid] = a[gid] + b[gid];
	}
	//printf("blockIdx.x: %d, blockDim.x: %d, threadIdx.x :%d, gid: %d, c: %d\n", blockIdx.x , blockDim.x , threadIdx.x,  gid, c[gid]);

}

void add_cpu(int* a, int* b, int* c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
		//printf("i: %d, c: %d \n", i, c[i]);
	}
	// printf("i: %d, c: %d \n", i, c[i]);
}

//void compare_arrays(int* a, int* b, int size)
//{
//	for (int i = 0; i < size; i++)
//	{
//		//printf("i: %d, a: %d, b: %d\n", i, a[i], b[i]);
//		if (a[i] != b[i])
//		{
//			//printf("i: %d, a: %d, b: %d\n", i, a[i], b[i]);
//			printf("Arrays are different \n");
//			return;
//		}
//	}
//	printf("Arrays are same \n");
//}

//int main()
//{
//	int array_size = 1000000;
//
//	int block_size = 1024;
//
//	cudaError error;	//intialize cuda error function,
//	int size_in_bytes = array_size * sizeof(int);
//
//	//host pointers
//	int* h_a, * h_b, * gpu_results, *cpu_results;
//
//	//memory allocation in host
//	h_a = (int*)malloc(size_in_bytes);	
//	h_b = (int*)malloc(size_in_bytes); 
//	gpu_results = (int*)malloc(size_in_bytes);
//	cpu_results = (int*)malloc(size_in_bytes);
//
//	//intialize host pointer
//	time_t t;
//	srand((unsigned)time(&t));
//
//	for (size_t i = 0; i < array_size; i++)
//	{
//		h_a[i] = (int)(rand() & 0xFF);
//	}
//	for (size_t i = 0; i < array_size; i++)
//	{
//		h_b[i] = (int)(rand() & 0xFF);
//	}
//
//	//remove garbadge value and intialize to 0
//	memset(gpu_results, 0, size_in_bytes);
//	memset(cpu_results, 0, size_in_bytes);
//
//	//Cpu function call, sum function
//	//clock_t cpu_start, cpu_end; //two clock variables to check execution time on cpu
//
//	auto cpu_start = std::chrono::high_resolution_clock::now();
//	add_cpu(h_a, h_b, cpu_results, array_size);
//	auto cpu_end = std::chrono::high_resolution_clock::now();
//
//	//device pointer
//	int* d_a, * d_b, * d_c;
//	error = cudaMalloc((int**)&d_a, size_in_bytes); //get return error from cuda. Use cuda error function
//	if (error != cudaSuccess)
//	{
//		fprintf(stderr, " Error : %s \n", cudaGetErrorString); //cuda get error from function
//	}
//	error = cudaMalloc((int**)&d_b, size_in_bytes);
//	if (error != cudaSuccess)
//	{
//		fprintf(stderr, " Error : %s \n", cudaGetErrorString); //cuda get error from function
//	}
//	error = cudaMalloc((int**)&d_c, size_in_bytes);
//	if (error != cudaSuccess)
//	{
//		fprintf(stderr, " Error : %s \n", cudaGetErrorString); //cuda get error from function
//	}
//
//	//Copy data
//	//clock_t htod_start, htod_end;
//	auto htod_start = std::chrono::high_resolution_clock::now();
//	cudaMemcpy(d_a, h_a, size_in_bytes, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_b, h_b, size_in_bytes, cudaMemcpyHostToDevice);
//	auto htod_end = std::chrono::high_resolution_clock::now();
//
//	//launching the grid
//	dim3 block(block_size);
//	dim3 grid((array_size / block.x) +1);
//
//	//clock_t gpu_start, gpu_end;
//	auto gpu_start = std::chrono::high_resolution_clock::now();
//	add << <grid, block >> > (d_a, d_b, d_c, array_size);
//	auto gpu_end = std::chrono::high_resolution_clock::now();
//
//	cudaDeviceSynchronize();
//
//	//copy results back to host
//	//clock_t dtoh_start, dtoh_end;
//	auto dtoh_start = std::chrono::high_resolution_clock::now();
//	cudaMemcpy(gpu_results, d_c, size_in_bytes, cudaMemcpyDeviceToHost);
//	auto dtoh_end = std::chrono::high_resolution_clock::now();
//
//
//	printf("Sum function execution time on CPU: %d micro sec \n", std::chrono::duration_cast<std::chrono::microseconds>(cpu_end - cpu_start).count());
//	printf("Sum function execution time on GPU: %d micro sec \n", std::chrono::duration_cast<std::chrono::microseconds>(gpu_end - gpu_start).count());
//	printf("Memory transfer from host to device, Upload time: %d micro sec \n", std::chrono::duration_cast<std::chrono::microseconds>(htod_end - htod_start).count());
//	printf("Memory transfer from device to host, Download time: %d micro sec \n", std::chrono::duration_cast<std::chrono::microseconds>(dtoh_end - dtoh_start).count());
//
//	//results comparison
//	compare_arrays(cpu_results, gpu_results, array_size);
//
//	//free cuda occupied memory
//	cudaFree(d_c);
//	cudaFree(d_b);
//	cudaFree(d_a);
//
//	//free host memory
//	free(gpu_results);
//	free(h_a);
//	free(h_b);
//
//	cudaDeviceReset();
//
//	std::cout << "Finished" << std::endl;
//	return 0;
//}