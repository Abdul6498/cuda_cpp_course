#include "hip/hip_runtime.h"
#include "Header.h"

__global__ void print_details_of_warps()
{
	int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

	int warp_id = threadIdx.x / 32;

	int gbid = blockIdx.y * gridDim.x + blockIdx.x;	//global block index

	printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, wrap_id : %d, gbid : %d \n",
		threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}

//int main()
//{
//	dim3 block_size(42);
//	dim3 grid_size(2, 2);
//
//	print_details_of_warps << <grid_size, block_size >> > ();
//	hipDeviceSynchronize();
//
//	hipDeviceReset();
//	printf("Finished \n");
//}